﻿#include	"Config.h"
#include	"mykernel.cuh"

void	CUDASetDevice();
void	OutputSpec(const hipDeviceProp_t);
FILE*	Open_file(string file_name,string file_name_exten);
void	inline	randomize()//random set up
{
	time_t t;
	srand((unsigned) time(&t));	
}
void	cudasafe( hipError_t error, char* message)
{
   if(error!=hipSuccess) { 
	   fprintf(stderr,"ERROR: %s : %i\n",message,error); 
	   const char *E = hipGetErrorString(hipGetLastError());
	   printf("%s\n", E);
	   system("pause");
	   exit(-1); 
   }
}

//Dimension
const	dim3	GRID(TUBE,1);							//dimension of grid
const	dim3	BLOCK(POPULATION,1);					//dimendion of block

int		main()
{
	//=====Set Up=====//
	CUDASetDevice();	//select device for cuda
	randomize();		//set up random for host
	string	E;			//error message		

	//=====Alocate Memory=====//
	//CUDA hiprand	
	hiprandState* devStates;												
	size_t	size_state		= (POPULATION);								
	size_t	mem_size_state	= sizeof(hiprandState) * TUBE * size_state;	
	hipMalloc((void **)&devStates, mem_size_state);cudasafe(hipGetLastError(),"Allocate hiprand");	

	dim3 D_curand(size_state,1);												
	/*setup_kernel<<<GRID,D>>>(devStates,rand());							
	hipDeviceSynchronize();
	cudasafe(hipGetLastError(),"set up hiprand");	*/

	//Populations	
	int*	d_popu;														
	size_t	size_popu		= (POPULATION * FIT_LENGTH * TUBE);			
	size_t	mem_size_popu	= sizeof(int) * size_popu;	
	hipMalloc((void **)&d_popu, mem_size_popu);cudasafe(hipGetLastError(),"Allocate Populations");
	hipMemset(d_popu,0,mem_size_popu);					

	//Scores
	double*	h_score;													
	double*	d_score;													
	size_t	size_score		= (POPULATION * TUBE);						
	size_t	mem_size_score	= sizeof(double) * size_popu;				
	h_score = (double*)malloc(mem_size_score);							
	hipMalloc((void **)&d_score, mem_size_score);cudasafe(hipGetLastError(),"Allocate Scores");
	hipMemset(d_score,0,mem_size_score);			

	//Fittness	
	double*	d_fit;														
	size_t	size_fit		= (POPULATION * TUBE);						
	size_t	mem_size_fit	= sizeof(double) * size_fit;					
	hipMalloc((void **)&d_fit, mem_size_fit);cudasafe(hipGetLastError(),"Allocate Fittness");							
	hipMemset(d_fit,0,mem_size_fit);	

	//Recording
	double*	h_rec;
	double*	d_rec;
	size_t	size_rec = (ITER * TUBE);
	size_t	mem_size_rec = sizeof(double) * size_rec;
	h_rec = (double*)malloc(mem_size_rec);
	hipMalloc((void **)&d_rec, mem_size_rec);cudasafe(hipGetLastError(),"Allocate Record");
	hipMemset(d_rec,0,mem_size_rec);	

	//Sample for Schema 	
	int*	d_smp_schema;												
	size_t	size_smp_schema		= (TUBE * FIT_LENGTH);	
	size_t	mem_size_smp_schema	= sizeof(int) * size_smp_schema;	
	hipMalloc((void **)&d_smp_schema, mem_size_smp_schema);cudasafe(hipGetLastError(),"Allocate Sample for Schema");
	hipMemset(d_smp_schema,0,mem_size_smp_schema);	

	//Sample for Enzyme	
	int*	d_smp_enzyme;
	size_t	size_smp_enzyme		= (TUBE * FIT_LENGTH);
	size_t	mem_size_smp_enzyme	= sizeof(int) * size_smp_enzyme;	
	hipMalloc((void **)&d_smp_enzyme, mem_size_smp_enzyme);cudasafe(hipGetLastError(),"Allocate Sample for Enzyme");
	hipMemset(d_smp_enzyme,0,mem_size_smp_enzyme);	

	//Restriction Enzyme 	
	double*	d_enzyme;
	size_t	size_enzyme			= (4 * FIT_LENGTH);
	size_t	mem_size_enzyme		= sizeof(double) * size_enzyme;
	hipMalloc((void **)&d_enzyme, mem_size_enzyme);cudasafe(hipGetLastError(),"Allocate Restriction Enzyme");
	hipMemset(d_enzyme,0,mem_size_enzyme);		

	//Restriction Enzyme Vector
	int*	d_enzyme_vec;
	size_t	size_enzyme_vec		= (FIT_LENGTH);
	size_t	mem_size_enzyme_vec = sizeof(int) * size_enzyme_vec;
	hipMalloc((void **)&d_enzyme_vec, mem_size_enzyme_vec);cudasafe(hipGetLastError(),"Allocate Restriction Enzyme vector");
	hipMemset(d_enzyme_vec,0,mem_size_enzyme_vec);

	//Global schema		
	double* d_g_schema;
	size_t	mem_size_schema = sizeof(double) * 4 * FIT_LENGTH;
	hipMalloc((void**) &d_g_schema,mem_size_schema);cudasafe(hipGetLastError(),"Allocate memory for schema");
	hipMemset(d_g_schema,0,mem_size_schema);

	//Local schema		
	double* d_l_schema;	
	size_t	mem_size_local_schema = sizeof(double) * 4 * FIT_LENGTH *TUBE;
	hipMalloc((void**) &d_l_schema,mem_size_local_schema);cudasafe(hipGetLastError(),"Allocate memory for schema");
	hipMemset(d_l_schema,0,mem_size_local_schema);

	//Search schema		
	double* d_s_schema;	
	size_t	mem_size_search_schema = sizeof(double) * 4 * FIT_LENGTH *TUBE;
	hipMalloc((void**) &d_s_schema,mem_size_search_schema);cudasafe(hipGetLastError(),"Allocate memory for schema");
	hipMemset(d_s_schema,0,mem_size_search_schema);

	//Entropy	
	double*	d_entropy;
	size_t	mem_size_entropy = sizeof(double) * TUBE;
	hipMalloc((void**) &d_entropy,mem_size_entropy);cudasafe(hipGetLastError(),"Allocate memory for Entropy");
	hipMemset(d_entropy,0,mem_size_entropy);

	//Open file	
	string file_ext_xls(" .xls");

	string file("EXL/CUDA DNA ");	
	FILE* f1 = Open_file(file,file_ext_xls);
	
	//=====Start Iteration=====//
	for(int simulat_loop=0;simulat_loop<SIMULATION;simulat_loop++){
		//cout<<simulat_loop<<endl;

		//=====Initialization=====//
		//=====Setup hiprand Kernel & Memory reset=====//
		setup_kernel<<<GRID,D_curand>>>(devStates,rand());
		hipMemset(d_popu,0,mem_size_popu);								
		hipMemset(d_score,0,mem_size_score);		
		hipMemset(d_fit,0,mem_size_fit);
		hipMemset(d_rec,0,mem_size_rec);
		hipMemset(d_smp_schema,0,mem_size_smp_schema);	
		hipMemset(d_smp_enzyme,0,mem_size_smp_enzyme);
		hipMemset(d_enzyme,0,mem_size_enzyme);
		hipMemset(d_enzyme_vec,0,mem_size_enzyme_vec);
		hipMemset(d_g_schema,0,mem_size_schema);
		hipMemset(d_l_schema,0,mem_size_local_schema);
		hipMemset(d_s_schema,0,mem_size_search_schema);
		hipMemset(d_entropy,0,mem_size_entropy);
		cudasafe(hipGetLastError(),"Setup hiprand Kernel & Memory reset");

		//=====Initial Population Kernel=====//
		Init_Popu_kernel<<<GRID,BLOCK>>>(d_popu,devStates);cudasafe(hipGetLastError(),"Initial Population Kernel");
		hipDeviceSynchronize();

		//=====Initial Enzyme Kernel=====//
		Init_Enzyme_kernel<<<1,1>>>(d_enzyme);cudasafe(hipGetLastError(),"Initial Enzyme Kernel");		
		
		cout<<endl;
		for(int I=0;I<ITER;I++){
			printf("\r%05.3f%%",(float)I/ITER*100);

			//=====Evaluation=====//
			Evaluation_kernel<<<GRID,BLOCK>>>(d_popu,d_score,d_fit);cudasafe(hipGetLastError(),"Evaluation Kernel");
			hipDeviceSynchronize();

			Popu_sort_kernel<<<1,GRID>>>(d_popu,d_score,d_fit);cudasafe(hipGetLastError(),"Population sorting Kernel");			
			Record_kernel<<<1,GRID>>>(d_score,d_rec,I);cudasafe(hipGetLastError(),"Record Kernel");

			//=====Extraction=====//
			Extract_kernel<<<1,GRID>>>(d_popu,d_smp_schema);cudasafe(hipGetLastError(),"Extract Kernel");

			//=====Updating Restriction Enzyme=====//			
			if(I==0){
				First_Filter_kernel<<<1,GRID>>>(d_popu,d_smp_enzyme);
			}else{
				Filter_kernel<<<1,1>>>(d_popu,d_smp_enzyme,OPT);
			}						
			cudasafe(hipGetLastError(),"Filter Kernel");

			Update_Enzyme_kernel<<<1,1>>>(d_smp_enzyme,d_enzyme,d_enzyme_vec);cudasafe(hipGetLastError(),"Update Enzyme Kernel");			

			//=====Updating Global Schema=====//
			Update_glo_schema_kernel<<<1,1>>>(d_smp_schema,d_g_schema,d_enzyme_vec);cudasafe(hipGetLastError(),"Update Global Schema");

			//=====Updating Local Schema=====//
			Update_loc_schema_kernel<<<1,1>>>(d_popu,d_l_schema,d_fit,d_enzyme_vec);cudasafe(hipGetLastError(),"Update Local Schema");
			
			//=====Updating Population=====//
			Update_population<<<GRID,BLOCK>>>(d_popu,d_g_schema,d_l_schema,d_s_schema,devStates);cudasafe(hipGetLastError(),"Update Population");
			hipDeviceSynchronize();

			Evaluation_kernel<<<GRID,BLOCK>>>(d_popu,d_score,d_fit);cudasafe(hipGetLastError(),"Evaluation Kernel");
			hipDeviceSynchronize();

			Popu_sort_kernel<<<1,GRID>>>(d_popu,d_score,d_fit);cudasafe(hipGetLastError(),"Population sorting Kernel");

			//=====Migration=====//			
			Get_Entropy<<<GRID,FIT_LENGTH>>>(d_s_schema,d_entropy);cudasafe(hipGetLastError(),"Get Entropy");
			//hipDeviceSynchronize();

			Migration<<<GRID,BLOCK>>>(d_entropy,d_popu,devStates);cudasafe(hipGetLastError(),"Migration");
			hipDeviceSynchronize();

		}//end of iteration loop
		//Print Result
		hipMemcpy(h_rec, d_rec, mem_size_rec, hipMemcpyDeviceToHost);
		cudasafe(hipGetLastError(),"cuda memory copy:rec");
				
		for(int i=0;i<ITER;i++){
			double	tmp[TUBE] = {};
			for(int j=0;j<TUBE;j++){		
				tmp[j]= *(h_rec+i*TUBE+j);
			}			
			
			if(OPT){
				fprintf(f1,"%.40f\t",Find_min(tmp,TUBE));
			}else{
				fprintf(f1,"%.40f\t",Find_max(tmp,TUBE));
			}
			
		}	
		fprintf(f1,"\n");
	}//end of simulation loop

	fclose(f1);

	//=====Release Memory=====//
	hipFree(devStates);
	hipFree(d_popu);
	hipFree(d_score);
	hipFree(d_fit);
	hipFree(d_rec);
	hipFree(d_smp_schema);
	hipFree(d_smp_enzyme);	
	hipFree(d_enzyme);
	hipFree(d_enzyme_vec);
	hipFree(d_g_schema);
	hipFree(d_l_schema);
	hipFree(d_s_schema);
	hipFree(d_entropy);
	free(h_score);
	free(h_rec);
	return 0;
}
void	CUDASetDevice()
{
	//Get number of devices
	int count;
	hipGetDeviceCount(&count);
	//Get device information
	 
	for( int i = 0; i < count; ++ i ){
		printf( "\n=== Device %i ===\n", i );
		hipDeviceProp_t  sDeviceProp;
		hipGetDeviceProperties( &sDeviceProp, i );
		OutputSpec( sDeviceProp );
	}
	cout<<endl;

	//Set device
	int dev;
	cout<<"Select Device:";
	cin>>dev;
	hipSetDevice(dev);
	//cout<<i<<endl;
}
void	OutputSpec( const hipDeviceProp_t sDevProp )
{
  printf( "Device name: %s\n", sDevProp.name );
  printf( "Device memory: %d\n", sDevProp.totalGlobalMem );
  printf( " Memory per-block: %d\n", sDevProp.sharedMemPerBlock );
  printf( " Register per-block: %d\n", sDevProp.regsPerBlock );
  printf( " Warp size: %d\n", sDevProp.warpSize );
  printf( " Memory pitch: %d\n", sDevProp.memPitch );
  printf( " Constant Memory: %d\n", sDevProp.totalConstMem );
  printf( "Max thread per-block: %d\n", sDevProp.maxThreadsPerBlock );
  printf( "Max thread dim: ( %d, %d, %d )\n", sDevProp.maxThreadsDim[0], sDevProp.maxThreadsDim[1], sDevProp.maxThreadsDim[2] );
  printf( "Max grid size: ( %d, %d, %d )\n", sDevProp.maxGridSize[0], sDevProp.maxGridSize[1], sDevProp.maxGridSize[2] );
  printf( "Ver: %d.%d\n", sDevProp.major, sDevProp.minor );
  printf( "Clock: %d\n", sDevProp.clockRate );
  printf( "textureAlignment: %d\n", sDevProp.textureAlignment );
  printf( "kernelExecTimeoutEnabled: %d\n", sDevProp.kernelExecTimeoutEnabled );
}

FILE*	Open_file(string file,string file_ext)
{
	tm* ptrnow;
	time_t loc_now = 0;
	time(&loc_now);
	ptrnow = localtime(&loc_now);
	string runtime = asctime(ptrnow);

	while(1){
		size_t found = runtime.find(":");
		if(found>=0 && found<runtime.size()){
			runtime[found] = '-';
		}else{
			break;
		}
	}
	
	string runtime2;
	for(int i=0;i<runtime.size()-1;i++){
		runtime2+=runtime[i];
	}

	file = file+runtime2+file_ext;
	FILE* f1 = fopen(file.c_str(), "w+");
	if(f1 == NULL){
		printf("error open %s file to write\n",file.c_str());
		system("pause");
		exit(1);
	}
	return f1;
}
